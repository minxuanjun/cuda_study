
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float* d_in, float* d_out)
{
    int id = threadIdx.x;
    d_out[id] = d_in[id]*d_in[id];
}


int main(int argc, char** argv){
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE*sizeof(float);

    float h_in[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];
    for(int i = 0; i < ARRAY_SIZE; i++){
        h_in[i] =i;
    }

    float* d_in;
    float* d_out;

    hipMalloc((void **)&d_in,ARRAY_BYTES);
    hipMalloc((void **)&d_out,ARRAY_BYTES);

    hipMemcpy(d_in,h_in,ARRAY_BYTES, hipMemcpyHostToDevice);

    square<<<1, ARRAY_BYTES>>>(d_in,d_out);
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for(int i=0; i < ARRAY_SIZE; i++)
    {
        printf("%f", h_out[i]);
        printf(((i%4)!=3) ? "\t" : "\n");
    }

    
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}